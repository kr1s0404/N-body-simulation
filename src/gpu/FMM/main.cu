#include "hip/hip_runtime.h"
/*
   Copyright 2023 Your Name

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

#include <iostream>
#include <cmath>
#include <opencv2/opencv.hpp>
#include "fastMultipoleCuda.cuh"
#include "constants.h"
#include "err.h"

// Constants for simulation
#define NUM_PARTICLES 10000
#define NUM_STEPS 1000
#define TIME_STEP 0.01f
#define RENDER_INTERVAL 10
#define DOMAIN_SIZE 10.0f
#define WINDOW_WIDTH 1024
#define WINDOW_HEIGHT 1024

// Function to initialize particles in a spiral pattern
void initSpiralParticles(Pos* positions, Vel* velocities, int numParticles) {
    float centerX = DOMAIN_SIZE / 2.0f;
    float centerY = DOMAIN_SIZE / 2.0f;
    float centerZ = DOMAIN_SIZE / 2.0f;
    
    // Central massive body
    positions[0].x = centerX;
    positions[0].y = centerY;
    positions[0].z = centerZ;
    positions[0].w = 1000.0f;  // Mass
    
    velocities[0].x = 0.0f;
    velocities[0].y = 0.0f;
    velocities[0].z = 0.0f;
    
    // Spiral particles
    for (int i = 1; i < numParticles; i++) {
        float angle = 0.1f * i;
        float radius = 0.1f + 0.01f * i;
        float height = 0.1f * sinf(angle * 0.1f);
        
        positions[i].x = centerX + radius * cosf(angle);
        positions[i].y = centerY + radius * sinf(angle);
        positions[i].z = centerZ + height;
        positions[i].w = 0.1f;  // Mass
        
        // Orbital velocity
        float speed = sqrtf(positions[0].w / radius) * 0.1f;
        velocities[i].x = -speed * sinf(angle);
        velocities[i].y = speed * cosf(angle);
        velocities[i].z = 0.0f;
    }
}

// Function to render particles to an image
void renderParticles(cv::Mat& image, Pos* positions, int numParticles) {
    // Clear image
    image = cv::Scalar(0, 0, 0);
    
    // Draw particles
    for (int i = 0; i < numParticles; i++) {
        float x = positions[i].x;
        float y = positions[i].y;
        
        // Map particle position to image coordinates
        int ix = (int)((x / DOMAIN_SIZE) * WINDOW_WIDTH);
        int iy = (int)((y / DOMAIN_SIZE) * WINDOW_HEIGHT);
        
        // Ensure coordinates are within image bounds
        if (ix >= 0 && ix < WINDOW_WIDTH && iy >= 0 && iy < WINDOW_HEIGHT) {
            // Color based on mass (brighter for more massive particles)
            int brightness = (int)(255.0f * fminf(positions[i].w / 1000.0f, 1.0f));
            
            // Draw particle
            if (i == 0) {
                // Central body (yellow)
                cv::circle(image, cv::Point(ix, iy), 10, cv::Scalar(0, brightness, brightness), -1);
            } else {
                // Other particles (blue)
                cv::circle(image, cv::Point(ix, iy), 2, cv::Scalar(brightness, 0, 0), -1);
            }
        }
    }
}

// Function to check command line arguments
bool checkArgs(int numParticles, int numSteps) {
    if (numParticles <= 0 || numSteps < 0) {
        std::cerr << "Invalid arguments. Usage: " << std::endl;
        std::cerr << "  ./FastMultipoleMethod [numParticles] [numSteps]" << std::endl;
        return false;
    }
    return true;
}

int main(int argc, char** argv) {
    // Initialize CUDA
    hipFree(0);
    
    // Parse command line arguments
    int numParticles = NUM_PARTICLES;
    int numSteps = NUM_STEPS;
    
    if (argc >= 3) {
        numParticles = atoi(argv[1]);
        numSteps = atoi(argv[2]);
    }
    
    if (!checkArgs(numParticles, numSteps)) {
        return -1;
    }
    
    std::cout << "Running simulation with " << numParticles << " particles for " 
              << numSteps << " steps" << std::endl;
    
    // Allocate host memory for particles
    Pos* h_pos = new Pos[numParticles];
    Vel* h_vel = new Vel[numParticles];
    
    // Initialize particles
    initSpiralParticles(h_pos, h_vel, numParticles);
    
    // Create FMM system
    FMMSystem* fmmSystem = createFMMSystem(numParticles, h_pos, h_vel);
    
    // Set domain size
    fmmSystem->setDomainSize(DOMAIN_SIZE);
    
    // Create video writer (headless mode)
    cv::VideoWriter video("nbody_fmm.avi", cv::VideoWriter::fourcc('M', 'J', 'P', 'G'), 30, 
                         cv::Size(WINDOW_WIDTH, WINDOW_HEIGHT));
    
    // Frame for rendering
    cv::Mat frame(WINDOW_HEIGHT, WINDOW_WIDTH, CV_8UC3);
    
    // Main simulation loop
    for (int step = 0; step < numSteps; step++) {
        // Perform simulation step
        fmmSystem->step(TIME_STEP);
        
        // Render every RENDER_INTERVAL steps
        if (step % RENDER_INTERVAL == 0) {
            // Get updated particle positions
            fmmSystem->getPositions(h_pos);
            
            // Render particles
            renderParticles(frame, h_pos, numParticles);
            
            // Write frame to video
            video.write(frame);
            
            // Print progress
            printf("Step %d/%d\n", step, numSteps);
        }
    }
    
    // Clean up
    video.release();
    std::cout << "Simulation complete. Video saved to nbody_fmm.avi" << std::endl;
    
    destroyFMMSystem(fmmSystem);
    delete[] h_pos;
    delete[] h_vel;
    
    return 0;
} 